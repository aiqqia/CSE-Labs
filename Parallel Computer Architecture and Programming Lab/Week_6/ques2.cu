#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Write a program in CUDA to read a sentence with equal length words. Count the number of times a given word is repeated in this sentence.  (Use Atomic function).  
// Sample string:  Pcap EEFM  exam Pcap test Pcap 
// Word: Pcap    Given word repeated 3 times 

__global__
void FindOccurrencesKernel(char *sentence, char *word, int *count, int length){
    int idx = threadIdx.x;
    int sum = 0;
    for (int i = 0; i < length; i++){
        if (word[i] == sentence[idx * length + i]){
            sum++;
        }
    }
    if (sum == length)
        atomicAdd(count, 1);
}
void FindOccurrences(char **sentence, char *word, int *count, int length, int n){
    char *d_sentence, *d_word;
    int *d_count;
    char s[n * length];
    int k = 0;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < length; j++){
            s[k++] = sentence[i][j];
        }
    }
    hipMalloc((void **)&d_sentence, sizeof(char) * length * n);
    hipMalloc((void **)&d_word, sizeof(char) * length);
    hipMalloc((void **)&d_count, sizeof(int));
    hipMemcpy(d_sentence, s, sizeof(char) * length * n, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(char) * length, hipMemcpyHostToDevice);
    FindOccurrencesKernel<<<1, n>>>(d_sentence, d_word, d_count, length);
    hipMemcpy(count, d_count, sizeof(int), hipMemcpyDeviceToHost);
}
int main(){
    printf("Enter the number of words in the sentence : ");
    int n;
    scanf("%d", &n);
    char **sentence = (char **)calloc(n, sizeof(char *));
    printf("Enter the equal word length : ");
    int length;
    scanf("%d", &length);
    printf("Enter the sentence : ");
    for (int i = 0; i < n; i++)
    {
        sentence[i] = (char *)calloc(length, sizeof(char));
        scanf(" %s", sentence[i]);
    }
    int count = 0;
    printf("Enter the word to be searched for : ");
    char *word = (char *)calloc(length, sizeof(char));
    scanf(" %s", word);
    FindOccurrences(sentence, word, &count, length, n);
    printf("The number of occurrences are : %d\n", count);
    return 0;
}