// 2. Write a program in CUDA to perform parallel Sparse Matrix - Vector Multiplication 
// using compressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format
//  in the host code.
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void SpMV_CSR(int num_rows, int* data, int* col_index, int* row_ptr, int* x, int* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        int dot = 0;
        int start = row_ptr[row];
        int stop = row_ptr[row + 1];
        for (int k = start; k < stop; k++) {
            dot += data[k] * x[col_index[k]];
        }
        y[row] = dot;
    }
}

int main() {
    int n, m;
    printf("Enter dimensions of matrix: ");
    scanf("%d%d", &n, &m);
    int* h_matrix = (int*)malloc(n * m * sizeof(int));
    int non_zero_count = 0;
    printf("Enter elements of matrix:\n");
    for (int i = 0; i < n * m; i++) {
        scanf("%d", h_matrix + i);
        non_zero_count += (h_matrix[i] != 0);
    }
    int* h_x = (int*)malloc(m * sizeof(int));
    printf("Enter %d elements of vector x: ", m);
    for (int i = 0; i < m; i++) {
        scanf("%d", h_x + i);
    }
    int* h_data = (int*)malloc(non_zero_count * sizeof(int));
    int* h_col_index = (int*)malloc(non_zero_count * sizeof(int));
    int* h_row_ptr = (int*)calloc(n + 1, sizeof(int));
    int* h_y = (int*)calloc(n, sizeof(int));
    int id = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            int k = i * m + j;
            if (h_matrix[k]) {
                h_data[id] = h_matrix[k];
                h_col_index[id] = j;
                id += 1;
            }
        }
        h_row_ptr[i + 1] = id;
    }

    int *d_data, *d_col_index, *d_row_ptr, *d_x, *d_y;
    hipMalloc((void**)&d_data, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_col_index, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (n + 1) * sizeof(int));
    hipMalloc((void**)&d_x, m * sizeof(int));
    hipMalloc((void**)&d_y, n * sizeof(int));

    hipMemcpy(d_data, h_data, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, h_col_index, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, m * sizeof(int), hipMemcpyHostToDevice);

    SpMV_CSR<<<1, n>>>(n, d_data, d_col_index, d_row_ptr, d_x, d_y);
    hipMemcpy(h_y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Y:");
    for (int i = 0; i < n; i++) {
        printf(" %d", h_y[i]);
    }
    printf("\n");
}