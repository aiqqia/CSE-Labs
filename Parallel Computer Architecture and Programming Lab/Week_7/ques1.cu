#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// 1. Write a CUDA program to perform convolution operation on one dimensional input array N of size width 
//    using a mask array M of size mask_width to produce the resultant one dimensional array P of size width 
//    using without and with constant Memory for Mask array.  
//    Add another kernel function to the first program to perform 1D convolution using shared memory. 
//    Find and display the time taken by both the kernels. 

#define MAX_MASK_WIDTH 5
__constant__ int M[MAX_MASK_WIDTH];

__global__
void oneDConvoluteKernel(int* d_A, int* d_M, int* d_P, int width, int mwidth){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    int c = 0;
    int sp = id - (mwidth/2);
    for(int j=0;j<mwidth;j++){
        if(sp+j >= 0 && sp+j < width)
            c += d_A[sp+j] * d_M[j];
    }
    d_P[id] = c;
}

__global__
void oneDConvoluteConstMemKernel(int* d_A, int* d_P, int width){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    int c = 0;
    int sp = id - (MAX_MASK_WIDTH/2);
    for(int j=0;j<MAX_MASK_WIDTH;j++){
        if(sp+j >= 0 && sp+j < width)
            c += d_A[sp+j] * M[j];
    }
    d_P[id] = c;
}

__global__
void oneDConvoluteSharedMemKernel(int *N, int *P, int mask_width, int width){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    extern __shared__ int N_shared[];
    // copy to shared memory
    N_shared[i] = N[i];
    __syncthreads();
    int Pvalue = 0;
    int N_start_point = i - (mask_width/2);
    for (int j = 0; j < mask_width; j++){
        if (N_start_point + j >= 0 && N_start_point + j < width)
            Pvalue += N_shared[N_start_point + j] * M[j];
    }
    P[i] = Pvalue;
}

__host__
void oneDConvolute(int* h_A, int* h_M, int* h_P, int width, int mwidth){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int *d_A, *d_M, *d_P;
    int size = width*sizeof(int);
    int msize = mwidth*sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_M, msize);
    hipMalloc((void**)&d_P, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, msize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, MAX_MASK_WIDTH * sizeof(int));
    hipEventRecord(start, 0);
    oneDConvoluteKernel<<<1,width>>>(d_A, d_M, d_P, width, mwidth);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    printf("The Convulated Array is : ");
    for(int i=0;i<width;i++){
        printf("%d ",h_P[i]);
    }
    printf("\nTotal Time Taken: %f\n", elapsedTime);
    hipEventRecord(start,0);
    oneDConvoluteConstMemKernel<<<1,width>>>(d_A,d_P,width);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    printf("The Convulated Array(Constant Memory) is : ");
    for(int i=0;i<width;i++){
        printf("%d ",h_P[i]);
    }
    printf("\nTotal Time Taken(Constant Memory): %f\n", elapsedTime);
    hipEventRecord(start,0);
    oneDConvoluteSharedMemKernel<<<1, width, width>>>(d_A,d_P,MAX_MASK_WIDTH,width);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    printf("The Convulated Array(Constant Memory) is : ");
    for(int i=0;i<width;i++){
        printf("%d ",h_P[i]);
    }
    printf("\nTotal Time Taken(Shared Memory): %f\n", elapsedTime);
    hipFree(d_A);
    hipFree(d_M);
    hipFree(d_P);
}

int main(){
    int *A, *M, *P;
    int width, mwidth;
    printf("Enter array width: ");
    scanf("%d", &width);
    int size = sizeof(int)*width;
    A = (int*)malloc(size);
    P = (int*)malloc(size);
    printf("Enter array elements: ");
    for(int i=0;i<width;i++){
        scanf("%d",&A[i]);
    }
    printf("Enter mask width(odd): ");
    scanf("%d",&mwidth);
    M = (int*)malloc(mwidth*sizeof(int));
    printf("Enter mask elements: ");
    for(int i=0;i<mwidth;i++){
        scanf("%d",&M[i]);
    }
    oneDConvolute(A,M,P,width,mwidth);
    return 0;
}