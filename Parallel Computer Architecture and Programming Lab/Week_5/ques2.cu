#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void selectionSortKernel(float* unsorted,float *sorted,int n){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	float key = unsorted[idx];
	int pos = 0;
	for(int i = 0;i < n;i++){
		if(unsorted[i] < key || (unsorted[i] == key && i < idx)){
			pos++;
		}
	}
	sorted[pos] = key;
}

void selectionSort(float *unsorted,float *sorted,int n){
	int size = n * sizeof(float);
	float *d_unsorted;
	float *d_sorted;

	hipMalloc((void**)&d_unsorted,size);
	hipMalloc((void**)&d_sorted,size);
	hipMemcpy(d_unsorted,unsorted,size,hipMemcpyHostToDevice);
	selectionSortKernel<<<1,n>>>(d_unsorted,d_sorted,n);
	hipMemcpy(sorted,d_sorted,size,hipMemcpyDeviceToHost);
	hipFree(d_unsorted);
	hipFree(d_sorted);
}
int main(){
	float *h_unsorted,*h_sorted;
	int n = 5;

	int size = n * sizeof(float);
	h_unsorted = (float*)malloc(size);
	h_sorted = (float*)malloc(size);

	for(int i = 0; i < 5; i++){
		h_unsorted[i] = rand()%50;
	}
	selectionSort(h_unsorted,h_sorted,n);
	printf("unsorted arr:");
	for(int i = 0;i < n;i++){
		printf("%f,",h_unsorted[i]);
	}
	printf("\n\n");
	printf("sorted arr:");
	for(int i = 0;i < n;i++){
		printf("%f,",h_sorted[i]);
	}
	printf("\n\n");	
	return 0;
}