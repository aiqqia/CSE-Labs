#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// 2. Write and execute a CUDA program to read an array of N integer values. Sort the array in parallel using parallel selection sort and store the result in another array.

__global__
void selectionSortKernel(int* unsorted,int *sorted,int n){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int key = unsorted[idx];
	int pos = 0;
	for(int i = 0;i < n;i++){
		if(unsorted[i] < key || (unsorted[i] == key && i < idx)){
			pos++;
		}
	}
	sorted[pos] = key;
}

void selectionSort(int *unsorted,int *sorted,int n){
	int size = n * sizeof(int);
	int *d_unsorted;
	int *d_sorted;

	hipMalloc((void**)&d_unsorted,size);
	hipMalloc((void**)&d_sorted,size);
	hipMemcpy(d_unsorted,unsorted,size,hipMemcpyHostToDevice);
	selectionSortKernel<<<1,n>>>(d_unsorted,d_sorted,n);
	hipMemcpy(sorted,d_sorted,size,hipMemcpyDeviceToHost);
	hipFree(d_unsorted);
	hipFree(d_sorted);
}
int main(){
	int *h_unsorted,*h_sorted;
	printf("Enter size(N): ");
    int n;
    scanf("%d",&n);
	int size = n * sizeof(int);
	h_unsorted = (int*)malloc(size);
	h_sorted = (int*)malloc(size);
    printf("\nEnter %d elements in unsorted array: ", n);
	for(int i = 0; i < n; i++){
		scanf("%d", &h_unsorted[i]);
	}
	selectionSort(h_unsorted,h_sorted,n);
	printf("unsorted arr:");
	for(int i = 0;i < n;i++){
		printf("%d,",h_unsorted[i]);
	}
	printf("\n\n");
	printf("sorted arr:");
	for(int i = 0;i < n;i++){
		printf("%d,",h_sorted[i]);
	}
	printf("\n\n");	
	return 0;
}