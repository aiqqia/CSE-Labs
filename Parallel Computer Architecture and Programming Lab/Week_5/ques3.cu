#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

__global__
void oddEven(int *arr,int n){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx%2 == 1 && idx + 1 < n){
		if(arr[idx] > arr[idx+1]){
			int temp = arr[idx];
			arr[idx] = arr[idx+1];
			arr[idx+1] = temp;
		}
	}
}

__global__
void evenOdd(int* arr,int n){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;

	if(idx%2 == 0 && idx + 1 < n){
		if(arr[idx] > arr[idx+1]){
			int temp = arr[idx];
			arr[idx] = arr[idx+1];
			arr[idx+1] = temp;
		}
	}
}

void oddEvenTranspositionSort(int *arr,int n){
	int size= n * sizeof(int);
	int *d_arr;
	hipMalloc((void**)&d_arr,size);
	hipMemcpy(d_arr,arr,size,hipMemcpyHostToDevice);
	for(int i = 0;i<=n/2;i++){
		oddEven<<<1,n>>>(d_arr,n);
		evenOdd<<<1,n>>>(d_arr,n);
	}
	hipMemcpy(arr,d_arr,size,hipMemcpyDeviceToHost);
	hipFree(d_arr);
}

int main(){
	int *h_arr;
	int n;
    printf("Enter size(N): ");
    scanf("%d", &n);
	int size = n * sizeof(int);
	h_arr = (int*)malloc(size);
    printf("Enter %d elements in unsorted array: ", n);
	for(int i = 0 ;i < n;i++){
		scanf("%d",&h_arr[i]);
	}
	printf("Unsorted Arr:");
	for(int i = 0 ;i < n;i++){
		printf("%d ",h_arr[i]);
	}
	printf("\n\n");
	oddEvenTranspositionSort(h_arr,n);
	printf("Sorted Arr: ");
	for(int i = 0 ;i < n;i++){
		printf("%d ",h_arr[i]);
	}
	printf("\n");
	return 0;
}
