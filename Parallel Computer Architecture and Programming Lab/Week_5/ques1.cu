#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// 1. Write and execute a program in CUDA to add two vectors of length N to meet the following requirements using 3 different kernels 

// a) block size as N 

// b) N threads within a block

// c) Keep the number of threads per block as 256 (constant) and vary the number of blocks to handle N elements.

__global__
void vecAddKernel_1a(float *a,float *b,float *c){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	c[idx] = a[idx]+b[idx];
}

__global__
void vecAddKernel_1b(float *a,float *b,float *c){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = a[idx] + b[idx];
}

__global__
void vecAddKernel_1c(float *a,float *b,float *c,int n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){
		c[idx] = a[idx] + b[idx];
	}
}

void vecAdd(float *a,float *b,float *c,int n){
	int size = n * sizeof(float);
	float *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_b,size);
	hipMalloc((void**) &d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%f,",a[i]);
	}
	printf("\n");
	printf("b:");
	for(int i = 0;i<n;i++){
		printf("%f,",b[i]);
	}
	printf("\n\n\n");
	vecAddKernel_1a<<<n,1>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1a):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");

	vecAddKernel_1b<<<1,n>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1b):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");	

	vecAddKernel_1c<<<((n+255)/256),256>>> (d_a,d_b,d_c,n);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1c):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(){
	float *ha,*hb,*hc;
	int n = 5;
	int size = n*sizeof(float);
	ha = (float*)malloc(size);
	hb = (float*)malloc(size);
	hc = (float*)malloc(size);

	for(int i = 0;i<n;i++){
		ha[i] = (i+1)*2;
		hb[i] = (i+1);
	}
	vecAdd(ha,hb,hc,n);
	return 0;
}