#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void vecAddKernel_1a(float *a,float *b,float *c){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	c[idx] = a[idx]+b[idx];
}

__global__
void vecAddKernel_1b(float *a,float *b,float *c){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = a[idx] + b[idx];
}

__global__
void vecAddKernel_1c(float *a,float *b,float *c,int n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){
		c[idx] = a[idx] + b[idx];
	}
}

void vecAdd(float *a,float *b,float *c,int n){
	int size = n * sizeof(float);
	float *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_b,size);
	hipMalloc((void**) &d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%f,",a[i]);
	}
	printf("\n");
	printf("b:");
	for(int i = 0;i<n;i++){
		printf("%f,",b[i]);
	}
	printf("\n\n\n");
	vecAddKernel_1a<<<n,1>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1a):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");

	vecAddKernel_1b<<<1,n>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1b):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");	

	vecAddKernel_1c<<<((n+255)/256),256>>> (d_a,d_b,d_c,n);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(from kernel 1c):");
	for(int i = 0;i<n;i++){
		printf("%f,",c[i]);
	}
	printf("\n\n");	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(){
	float *ha,*hb,*hc;
	int n = 5;
	int size = n*sizeof(float);
	ha = (float*)malloc(size);
	hb = (float*)malloc(size);
	hc = (float*)malloc(size);

	for(int i = 0;i<n;i++){
		ha[i] = (i+1)*2;
		hb[i] = (i+1);
	}
	vecAdd(ha,hb,hc,n);
	return 0;
}